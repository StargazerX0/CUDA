#include "hip/hip_runtime.h"
// optimized_kernel_firstMatch.cu

#include <hip/hip_runtime.h>
#include <vector>
#include <string>
#include <iostream>
#include "kseq/kseq.h"
#include "common.h"


// CUDA kernel to find the first match for each sample-signature pair
__global__ void matchKernelFirstMatch(
    const char* __restrict__ d_samples,
    const int* __restrict__ d_sample_lengths,
    const size_t* __restrict__ d_sample_offsets,
    const char* __restrict__ d_signatures,
    const int* __restrict__ d_sig_lengths,
    const size_t* __restrict__ d_sig_offsets,
    const char* __restrict__ d_quals,
    double* d_match_scores,
    int num_samples,
    int num_signatures)
{
    // Calculate the unique pair index for each thread in the x-dimension
    int pair_x = blockIdx.x * blockDim.x + threadIdx.x;
    if (pair_x >= num_samples * num_signatures) return;

    // Determine the sample and signature indices
    int sample_idx = pair_x / num_signatures;
    int signature_idx = pair_x % num_signatures;

    // Retrieve lengths and offsets
    int sample_length = d_sample_lengths[sample_idx];
    int signature_length = d_sig_lengths[signature_idx];

    // If signature is longer than sample, no match is possible
    if (signature_length > sample_length) {
        d_match_scores[pair_x] = 0.0;
        return;
    }

    // Pointers to the sample, signature, and quality strings
    const char* sample = &d_samples[d_sample_offsets[sample_idx]];
    const char* signature = &d_signatures[d_sig_offsets[signature_idx]];
    const char* qual = &d_quals[d_sample_offsets[sample_idx]];

    int num_positions = sample_length - signature_length + 1;

    // Shared memory for the earliest match index per pair in the block
    __shared__ int first_match_indices[16]; // Assuming blockDim.x = 16

    // Initialize shared memory (only once per pair)
    if (threadIdx.y == 0) {
        first_match_indices[threadIdx.x] = num_positions; // Initialize to max
    }
    __syncthreads();

    // Each thread in the y-dimension handles different starting positions
    for (int pos = threadIdx.y; pos < num_positions; pos += blockDim.y) {
        bool match = true;

        // Check for a match at the current position
        for (int j = 0; j < signature_length; ++j) {
            char s = sample[pos + j];
            char sig = signature[j];

            // Handle 'N' wildcard in sample or signature
            if ((s != 'N' && sig != 'N' && s != sig)) {
                match = false;
                break;
            }
        }

        if (match) {
            // Atomically update the first match index if a match is found
            atomicMin(&first_match_indices[threadIdx.x], pos);
        }
    }

    __syncthreads();

    // Only one thread computes the confidence score per pair
    if (threadIdx.y == 0) {
        int first_match = first_match_indices[threadIdx.x];
        if (first_match < num_positions) {
            // Compute confidence score for the first match
            double confidence_sum = 0.0f;
            for (int j = 0; j < signature_length; ++j) {
                int phred = static_cast<int>(qual[first_match + j]) - 33;
                confidence_sum += static_cast<double>(phred);
            }
            double avg_confidence = confidence_sum / static_cast<double>(signature_length);
            d_match_scores[pair_x] = avg_confidence;
        } else {
            // No match found
            d_match_scores[pair_x] = 0.0f;
        }
    }
}

void runMatcher(const std::vector<klibpp::KSeq>& samples,
               const std::vector<klibpp::KSeq>& signatures,
               std::vector<MatchResult>& matches) {

    // Number of samples and signatures
    int num_samples = samples.size();
    int num_signatures = signatures.size();

    if (num_samples == 0 || num_signatures == 0) {
        return;
    }

    // Calculate total lengths and prepare offsets
    size_t total_sample_length = 0;
    size_t total_sig_length = 0;

    std::vector<size_t> h_sample_offsets(num_samples);
    std::vector<int> h_sample_lengths(num_samples);
    size_t offset = 0;
    for (int i = 0; i < num_samples; ++i) {
        h_sample_lengths[i] = samples[i].seq.length();
        h_sample_offsets[i] = offset;
        total_sample_length += samples[i].seq.length();
        offset += samples[i].seq.length();
    }

    std::vector<int> h_sig_lengths(num_signatures);
    std::vector<size_t> h_sig_offsets(num_signatures);
    offset = 0;
    for (int i = 0; i < num_signatures; ++i) {
        h_sig_lengths[i] = signatures[i].seq.length();
        h_sig_offsets[i] = offset;
        total_sig_length += signatures[i].seq.length();
        offset += signatures[i].seq.length();
    }

    // Allocate host memory for samples, signatures, and qualities
    std::vector<char> h_samples_data(total_sample_length);
    std::vector<char> h_signatures_data(total_sig_length);
    std::vector<char> h_quals_data(total_sample_length);

    // Populate host memory
    for (int i = 0; i < num_samples; ++i) {
        memcpy(&h_samples_data[h_sample_offsets[i]], samples[i].seq.c_str(), h_sample_lengths[i]);
        memcpy(&h_quals_data[h_sample_offsets[i]], samples[i].qual.c_str(), h_sample_lengths[i]);
    }
    for (int i = 0; i < num_signatures; ++i) {
        memcpy(&h_signatures_data[h_sig_offsets[i]], signatures[i].seq.c_str(), h_sig_lengths[i]);
    }

    // Allocate device memory
    char* d_samples;
    char* d_signatures;
    char* d_quals;
    double* d_match_scores;

    int* d_sample_lengths;
    int* d_sig_lengths;

    size_t* d_sample_offsets;
    size_t* d_sig_offsets;

    hipMalloc(&d_samples, total_sample_length * sizeof(char));
    hipMalloc(&d_signatures, total_sig_length * sizeof(char));
    hipMalloc(&d_quals, total_sample_length * sizeof(char));
    hipMalloc(&d_match_scores, num_samples * num_signatures * sizeof(double));

    hipMalloc(&d_sample_lengths, num_samples * sizeof(int));
    hipMalloc(&d_sig_lengths, num_signatures * sizeof(int));

    hipMalloc(&d_sample_offsets, num_samples * sizeof(size_t));
    hipMalloc(&d_sig_offsets, num_signatures * sizeof(size_t));

    // Copy data from host to device
    hipMemcpy(d_samples, h_samples_data.data(), total_sample_length * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_signatures, h_signatures_data.data(), total_sig_length * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_quals, h_quals_data.data(), total_sample_length * sizeof(char), hipMemcpyHostToDevice);

    hipMemcpy(d_sample_lengths, h_sample_lengths.data(), num_samples * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_sig_lengths, h_sig_lengths.data(), num_signatures * sizeof(int), hipMemcpyHostToDevice);

    hipMemcpy(d_sample_offsets, h_sample_offsets.data(), num_samples * sizeof(size_t), hipMemcpyHostToDevice);
    hipMemcpy(d_sig_offsets, h_sig_offsets.data(), num_signatures * sizeof(size_t), hipMemcpyHostToDevice);

    // Define kernel launch parameters
    dim3 threadsPerBlock(16, 16); // 16x16 threads per block
    int total_pairs = num_samples * num_signatures;
    int blocksPerGrid = (total_pairs + threadsPerBlock.x - 1) / threadsPerBlock.x;

    // Initialize match scores to zero
    hipMemset(d_match_scores, 0, num_samples * num_signatures * sizeof(double));

    // Launch the optimized first-match kernel
    matchKernelFirstMatch<<<blocksPerGrid, threadsPerBlock>>>(
        d_samples, d_sample_lengths, d_sample_offsets,
        d_signatures, d_sig_lengths, d_sig_offsets,
        d_quals, d_match_scores,
        num_samples, num_signatures
    );

    // Check for kernel launch errors
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "Kernel launch failed: " << hipGetErrorString(err) << std::endl;
        // Handle error as needed
    }

    // Allocate host memory for match scores
    std::vector<double> h_match_scores(num_samples * num_signatures);

    // Wait for GPU to finish
    hipDeviceSynchronize();

    // Copy match scores back to host
    hipMemcpy(h_match_scores.data(), d_match_scores, num_samples * num_signatures * sizeof(double), hipMemcpyDeviceToHost);

    // Populate the matches vector
    matches.reserve(num_samples * num_signatures); // Reserve space to avoid reallocations
    for (int i = 0; i < num_samples; ++i) {
        for (int j = 0; j < num_signatures; ++j) {
            double score = h_match_scores[i * num_signatures + j];
            if (score > 0.0f) { // A match is indicated by score > 0
                MatchResult result;
                result.sample_name = samples[i].name;
                result.signature_name = signatures[j].name;
                result.match_score = static_cast<double>(score);
                matches.push_back(result);
            }
        }
    }

    // Free device memory
    hipFree(d_samples);
    hipFree(d_signatures);
    hipFree(d_quals);
    hipFree(d_match_scores);
    hipFree(d_sample_lengths);
    hipFree(d_sig_lengths);
    hipFree(d_sample_offsets);
    hipFree(d_sig_offsets);
}
